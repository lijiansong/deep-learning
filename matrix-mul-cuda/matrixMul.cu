#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */

/* Matrix multiplication: C = A * B.
 * Host code.
 *
 * This sample implements matrix multiplication and is exactly the same as
 * Chapter 7 of the programming guide.
 * It has been written for clarity of exposition to illustrate various CUDA
 * programming principles, not with the goal of providing the most
 * performant generic kernel for matrix multiplication.
 *
 * CUBLAS provides high-performance matrix multiplication.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
//#include <cutil_inline.h>
//#include <helper_functions.h>

// includes, kernels
#include <matrixMul_kernel.cuh>
#include <matrixMul_cublas.cuh>
#include <matrixMul_naive.cuh>
#include <matrixMul_tiling.cuh>
#include <matrixMul_coalescing.cuh>
#include <matrixMul_noBankConflict.cuh>
#include <matrixMul_compOpt.cuh>
#include <matrixMul_unroll.cuh>
#include <matrixMul_prefetch.cuh>

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char** argv);
void randomInit(float*, int);
void printDiff(float*, float*, int, int);

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int, unsigned int);

////////////////////////////////////////////////////////////////////////////////
// Helper Functions
////////////////////////////////////////////////////////////////////////////////
#ifndef STRNCASECMP
#define STRNCASECMP strncasecmp
#endif

inline int stringRemoveDelimiter(char delimiter, const char *string)
{
    int string_start = 0;

    while (string[string_start] == delimiter)
    {
        string_start++;
    }

    if (string_start >= (int)strlen(string)-1)
    {
        return 0;
    }

    return string_start;
}

inline bool checkCmdLineFlag(const int argc, const char **argv, const char *string_ref)
{
    bool bFound = false;

    if (argc >= 1)
    {
        for (int i=1; i < argc; i++)
        {
            int string_start = stringRemoveDelimiter('-', argv[i]);
            const char *string_argv = &argv[i][string_start];

            const char *equal_pos = strchr(string_argv, '=');
            int argv_length = (int)(equal_pos == 0 ? strlen(string_argv) : equal_pos - string_argv);

            int length = (int)strlen(string_ref);

            if (length == argv_length && !STRNCASECMP(string_argv, string_ref, length))
            {
                bFound = true;
                continue;
            }
        }
    }

    return bFound;
}

inline int getCmdLineArgumentInt(const int argc, const char **argv, const char *string_ref)
{
    bool bFound = false;
    int value = -1;

    if (argc >= 1)
    {
        for (int i=1; i < argc; i++)
        {
            int string_start = stringRemoveDelimiter('-', argv[i]);
            const char *string_argv = &argv[i][string_start];
            int length = (int)strlen(string_ref);

            if (!STRNCASECMP(string_argv, string_ref, length))
            {
                if (length+1 <= (int)strlen(string_argv))
                {
                    int auto_inc = (string_argv[length] == '=') ? 1 : 0;
                    value = atoi(&string_argv[length + auto_inc]);
                }
                else
                {
                    value = 0;
                }

                bFound = true;
                continue;
            }
        }
    }

    if (bFound)
    {
        return value;
    }
    else
    {
        return 0;
    }
}


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv)
{
    runTest(argc, argv);

    exit(EXIT_SUCCESS);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char** argv)
{

    /****************************************************/
    /*  Preparations                                    */
    /****************************************************/

    printf("[Matrix Multiply Using CUDA] - Starting...\n");

    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
        checkCmdLineFlag(argc, (const char **)argv, "?"))
    {
        printf("Usage -device=n (n >= 0 for deviceID)\n");
        printf("      -wA=WidthA -hA=HeightA (Width x Height of Matrix A)\n");
        printf("      -wB=WidthB -hB=HeightB (Width x Height of Matrix B)\n");
        printf("  Note: Outer matrix dimensions of A & B matrices must be equal.\n");

        exit(EXIT_SUCCESS);
    }

    // By default, we use device 0, otherwise we override the device ID based on what is provided at the command line
    int devID = 0;

    if (checkCmdLineFlag(argc, (const char **)argv, "device"))
    {
        devID = getCmdLineArgumentInt(argc, (const char **)argv, "device");
        hipSetDevice(devID);
    }

    hipError_t error;
    hipDeviceProp_t deviceProp;
    error = hipGetDevice(&devID);

    if (error != hipSuccess)
    {
        printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
    }

    error = hipGetDeviceProperties(&deviceProp, devID);

    if (deviceProp.computeMode == hipComputeModeProhibited)
    {
        fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
        exit(EXIT_SUCCESS);
    }

    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
    }
    else
    {
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    }

    // utilities
    hipEvent_t start;
    hipEvent_t stop;
    float msecTotal;

    // set seed for rand()
    srand(2006);

    // allocate host memory for matrices A and B
    unsigned int size_A = WA * HA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*) malloc(mem_size_A);
    unsigned int size_B = WB * HB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*) malloc(mem_size_B);
    float flop = 2 * (float)WC * (float)HC * (float)WA;

    // initialize host memory
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);
    
    // allocate device memory
    float* d_A;
    hipMalloc((void**) &d_A, mem_size_A);
    float* d_B;
    hipMalloc((void**) &d_B, mem_size_B);

    // allocate device memory for result
    unsigned int size_C = WC * HC;
    unsigned int mem_size_C = sizeof(float) * size_C;
    float* d_C;
    hipMalloc((void**) &d_C, mem_size_C);

    // allocate host memory for the result
    float* h_C = (float*) malloc(mem_size_C);

#if CHECK_RESULT == 1
    // create and start timer
    hipEventCreate(&start);
    hipEventRecord(start, NULL); 
    // compute reference solution
    float* reference = (float*) malloc(mem_size_C);
    computeGold(reference, h_A, h_B, HA, WA, WB);
    // stop and destroy timer
    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);
    printf("Naive CPU (Golden Reference)\n");
    printf("Processing time: %f (ms), GFLOPS: %f \n", msecTotal, flop / msecTotal/ 1e+6);
#endif

    dim3 threads,grid;

    /****************************************************/
    /*  CUDA SDK example                                */
    /****************************************************/

    // create and start timer
    hipEventCreate(&start);
    hipEventRecord(start, NULL);
    // copy host memory to device
    hipMemcpy(d_A, h_A, mem_size_A,
                              hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_B,
                              hipMemcpyHostToDevice);
   // setup execution parameters
    threads = dim3(BLOCK_SIZE, BLOCK_SIZE);
    grid = dim3(WC / threads.x, HC / threads.y);
    // execute the kernel
    matrixMul<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
    // copy result from device to host
    hipMemcpy(h_C, d_C, mem_size_C,
                              hipMemcpyDeviceToHost);
    // stop and destroy timer
    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);
    printf("GPU SDK Sample\n");
    printf("Processing time: %f (ms), GFLOPS: %f \n", msecTotal, flop / msecTotal/ 1e+6);
#if CHECK_RESULT == 1
    // check result
    printDiff(reference, h_C, WC, HC);
#endif

    /****************************************************/
    /*  naive implementation on GPU                     */
    /****************************************************/

#if ENABLE_NAIVE == 1

    // create and start timer
    hipEventCreate(&start);
    hipEventRecord(start, NULL);
    // setup execution parameters
    threads = dim3(BLOCK_SIZE, BLOCK_SIZE);
    grid = dim3(WC / threads.x, HC / threads.y);
    // copy host memory to device
    hipMemcpy(d_A, h_A, mem_size_A,
                              hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_B,
                              hipMemcpyHostToDevice);
    // naive implementation
    matrixMul_naive<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
    // copy result from device to host
    hipMemcpy(h_C, d_C, mem_size_C,
                              hipMemcpyDeviceToHost);
    // stop and destroy timer
    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);
    printf("Naive GPU\n");
    printf("Processing time: %f (ms), GFLOPS: %f \n", msecTotal, flop / msecTotal/ 1e+6);
#if CHECK_RESULT == 1
    // check result
    printDiff(reference, h_C, WC, HC);
#endif

#endif

    /****************************************************/
    /*  Tiling without global mem coalescing            */
    /****************************************************/

    // create and start timer
    hipEventCreate(&start);
    hipEventRecord(start, NULL);
    // setup execution parameters
    threads = dim3(BLOCK_SIZE, BLOCK_SIZE);
    grid = dim3(WC / threads.x, HC / threads.y);
    // copy host memory to device
    hipMemcpy(d_A, h_A, mem_size_A,
                              hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_B,
                              hipMemcpyHostToDevice);
    // naive implementation
    matrixMul_tiling<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
    // copy result from device to host
    hipMemcpy(h_C, d_C, mem_size_C,
                              hipMemcpyDeviceToHost);
    // stop and destroy timer
    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);
    printf("Tiling GPU\n");
    printf("Processing time: %f (ms), GFLOPS: %f \n", msecTotal, flop / msecTotal/ 1e+6);
#if CHECK_RESULT == 1
    // check result
    printDiff(reference, h_C, WC, HC);
#endif

    /****************************************************/
    /*  Global mem coalescing with smem bank conflict   */
    /****************************************************/

    // create and start timer
    hipEventCreate(&start);
    hipEventRecord(start, NULL); 
    // setup execution parameters
    threads = dim3(BLOCK_SIZE, BLOCK_SIZE);
    grid = dim3(WC / threads.x, HC / threads.y);
    // copy host memory to device
    hipMemcpy(d_A, h_A, mem_size_A,
                              hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_B,
                              hipMemcpyHostToDevice);
    // naive implementation
    matrixMul_coalescing<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
    // copy result from device to host
    hipMemcpy(h_C, d_C, mem_size_C,
                              hipMemcpyDeviceToHost);
    // stop and destroy timer
    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);
    printf("Global mem coalescing GPU\n");
    printf("Processing time: %f (ms), GFLOPS: %f \n", msecTotal, flop / msecTotal/ 1e+6);
#if CHECK_RESULT == 1
    // check result
    printDiff(reference, h_C, WC, HC);
#endif

    /****************************************************/
    /*  Global mem coalescing w/o smem bank conflict    */
    /****************************************************/

    // create and start timer
    hipEventCreate(&start);
    hipEventRecord(start, NULL); 
    // setup execution parameters
    threads = dim3(BLOCK_SIZE, BLOCK_SIZE);
    grid = dim3(WC / threads.x, HC / threads.y);
    // copy host memory to device
    hipMemcpy(d_A, h_A, mem_size_A,
                              hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_B,
                              hipMemcpyHostToDevice);
    // naive implementation
    matrixMul_noBankConflict<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
    // copy result from device to host
    hipMemcpy(h_C, d_C, mem_size_C,
                              hipMemcpyDeviceToHost);
    // stop and destroy timer
    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);
    printf("Remove shared mem bank conflict GPU\n");
    printf("Processing time: %f (ms), GFLOPS: %f \n", msecTotal, flop / msecTotal/ 1e+6);
#if CHECK_RESULT == 1
    // check result
    printDiff(reference, h_C, WC, HC);
#endif

    /****************************************************/
    /*  Threads perform computation optimizatin         */
    /****************************************************/

    // create and start timer
    hipEventCreate(&start);
    hipEventRecord(start, NULL); 
    // setup execution parameters
    threads = dim3(BLOCK_SIZE, 4);
    grid = dim3(WC / (BLOCK_SIZE*4), HC / BLOCK_SIZE);
    // copy host memory to device
    hipMemcpy(d_A, h_A, mem_size_A,
                              hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_B,
                              hipMemcpyHostToDevice);
    // naive implementation
    matrixMul_compOpt<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
    // copy result from device to host
    hipMemcpy(h_C, d_C, mem_size_C,
                              hipMemcpyDeviceToHost);
    // stop and destroy timer
    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);
    printf("Threads perform computation optimization GPU\n");
    printf("Processing time: %f (ms), GFLOPS: %f \n", msecTotal, flop / msecTotal/ 1e+6);
#if CHECK_RESULT == 1
    // check result
    printDiff(reference, h_C, WC, HC);
#endif


    /****************************************************/
    /*  Loop Unrolling                                  */
    /****************************************************/

    // create and start timer
    hipEventCreate(&start);
    hipEventRecord(start, NULL); 
    // setup execution parameters
    threads = dim3(BLOCK_SIZE, 4);
    grid = dim3(WC / (BLOCK_SIZE*4), HC / BLOCK_SIZE);
    // copy host memory to device
    hipMemcpy(d_A, h_A, mem_size_A,
                              hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_B,
                              hipMemcpyHostToDevice);
    // naive implementation
    matrixMul_unroll<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
    // copy result from device to host
    hipMemcpy(h_C, d_C, mem_size_C,
                              hipMemcpyDeviceToHost);
    // stop and destroy timer
    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);
    printf("Loop unrolling GPU\n");
    printf("Processing time: %f (ms), GFLOPS: %f \n", msecTotal, flop / msecTotal/ 1e+6);
#if CHECK_RESULT == 1
    // check result
    printDiff(reference, h_C, WC, HC);
#endif

    /****************************************************/
    /*  Prefetching                                     */
    /****************************************************/

    // create and start timer
    hipEventCreate(&start);
    hipEventRecord(start, NULL); 
    // setup execution parameters
    threads = dim3(BLOCK_SIZE, 4);
    grid = dim3(WC / (BLOCK_SIZE*4), HC / BLOCK_SIZE);
    // copy host memory to device
    hipMemcpy(d_A, h_A, mem_size_A,
                              hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_B,
                              hipMemcpyHostToDevice);
    // naive implementation
    matrixMul_prefetch<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
    // copy result from device to host
    hipMemcpy(h_C, d_C, mem_size_C,
                              hipMemcpyDeviceToHost);
    // stop and destroy timer
    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);
    printf("Prefetching GPU\n");
    printf("Processing time: %f (ms), GFLOPS: %f \n", msecTotal, flop / msecTotal/ 1e+6);
#if CHECK_RESULT == 1
    // check result
    printDiff(reference, h_C, WC, HC);
#endif

    /****************************************************/
    /*  Cleaning                                        */
    /****************************************************/

    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
#if CHECK_RESULT == 1
    free(reference);
#endif
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipDeviceReset();
}

// Allocates a matrix with random float entries.
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

void printDiff(float *data1, float *data2, int width, int height)
{
  int i,j,k;
  int error_count=0;
  for (j=0; j<height; j++) {
    for (i=0; i<width; i++) {
      k = j*width+i;
      if (fabs(data1[k] - data2[k]) > 0.1 ) {
         printf("diff(%d,%d) CPU=%4.4f, GPU=%4.4f \n", i,j, data1[k], data2[k]);
         error_count++;
      }
    }
  }
  printf("Total Errors = %d \n", error_count);
}

